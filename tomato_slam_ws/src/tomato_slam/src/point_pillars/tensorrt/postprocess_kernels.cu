
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>

namespace tomato_slam {

constexpr int THREADS_PER_BLOCK = 256;

// 设备常量
__constant__ float score_threshold;
__constant__ int num_classes;
__constant__ int num_dir_bins;
__constant__ float dir_offset;
__constant__ float dir_limit_offset;
__constant__ float nms_threshold;
__constant__ int max_detections;

// 筛选检测结果
__global__ void filter_detections_kernel(const float* box_preds, 
                                       const float* cls_preds, 
                                       const float* dir_preds,
                                       int num_boxes,
                                       float* filtered_boxes,
                                       float* filtered_scores,
                                       float* filtered_dirs,
                                       int* num_filtered) {
    int box_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (box_idx >= num_boxes) return;

    // 获取当前框的类别得分
    float score = 0.0f;
    int class_idx = 0;
    for (int c = 0; c < num_classes; ++c) {
        float cls_score = cls_preds[box_idx * num_classes + c];
        if (cls_score > score) {
            score = cls_score;
            class_idx = c;
        }
    }
    
    // 应用置信度阈值
    if (score < score_threshold) return;
    
    // 获取方向预测
    float dir_score_0 = dir_preds[box_idx * num_dir_bins + 0];
    float dir_score_1 = dir_preds[box_idx * num_dir_bins + 1];
    int dir_label = dir_score_0 > dir_score_1 ? 0 : 1;
    
    // 添加到过滤后的结果
    int idx = atomicAdd(num_filtered, 1);
    if (idx < max_detections) {
        // 复制框预测 (7个值: x, y, z, w, l, h, yaw)
        for (int i = 0; i < 7; ++i) {
            filtered_boxes[idx * 7 + i] = box_preds[box_idx * 7 + i];
        }
        
        // 如果方向标签为1，则旋转180度
        if (dir_label == 1) {
            float yaw = filtered_boxes[idx * 7 + 6];
            filtered_boxes[idx * 7 + 6] = yaw + 3.14159265f;
        }
        
        // 存储得分和类别
        filtered_scores[idx] = score;
        filtered_dirs[idx] = dir_label;
    }
}

// 执行后处理（筛选和方向修正）
int filterDetections(const float* box_preds, 
                    const float* cls_preds, 
                    const float* dir_preds,
                    int num_boxes,
                    float* filtered_boxes,
                    float* filtered_scores,
                    float* filtered_dirs,
                    int* dev_num_filtered) {
    // 初始化检测数量为0
    hipMemset(dev_num_filtered, 0, sizeof(int));
    
    // 计算并设置启动配置
    int num_blocks = (num_boxes + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    
    // 调用筛选核函数
    filter_detections_kernel<<<num_blocks, THREADS_PER_BLOCK>>>(
        box_preds, cls_preds, dir_preds, num_boxes,
        filtered_boxes, filtered_scores, filtered_dirs, dev_num_filtered);
    
    // 同步
    hipDeviceSynchronize();
    
    // 获取过滤后的检测数量
    int num_filtered = 0;
    hipMemcpy(&num_filtered, dev_num_filtered, sizeof(int), hipMemcpyDeviceToHost);
    
    return num_filtered;
}

// 设置后处理参数
void setPostprocessParams(float score_thresh, int num_cls, int num_dir_bin,
                         float dir_off, float dir_limit_off, float nms_thresh, int max_det) {
    hipMemcpyToSymbol(HIP_SYMBOL(score_threshold), &score_thresh, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(num_classes), &num_cls, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(num_dir_bins), &num_dir_bin, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(dir_offset), &dir_off, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(dir_limit_offset), &dir_limit_off, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(nms_threshold), &nms_thresh, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(max_detections), &max_det, sizeof(int));
}

} // namespace tomato_slam
