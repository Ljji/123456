// ~/tomato_slam_ws/src/tomato_slam/src/point_pillars/tensorrt/preprocess_kernels.cu
#include <hip/hip_runtime.h>

namespace tomato_slam {

// 声明核函数
__global__ void preprocessPointsKernel(const float* points, int num_points, 
                                      float* voxel_features, int* voxel_num_points,
                                      float min_x, float min_y, float min_z,
                                      float max_x, float max_y, float max_z,
                                      float voxel_x, float voxel_y, float voxel_z,
                                      int grid_x, int grid_y, int grid_z,
                                      int max_points_per_voxel, int feature_num) {
    // 实现点云预处理逻辑
    // ...
}

// 主机端函数包装
void launchPreprocessKernel(const float* points, int num_points,
                          float* voxel_features, int* voxel_num_points,
                          const float* voxel_params, const int* grid_params, 
                          int max_points_per_voxel, int feature_num,
                          hipStream_t stream) {
    
    // 提取参数
    float min_x = voxel_params[0];
    float min_y = voxel_params[1];
    float min_z = voxel_params[2];
    float max_x = voxel_params[3];
    float max_y = voxel_params[4];
    float max_z = voxel_params[5];
    float voxel_x = voxel_params[6];
    float voxel_y = voxel_params[7];
    float voxel_z = voxel_params[8];
    
    int grid_x = grid_params[0];
    int grid_y = grid_params[1];
    int grid_z = grid_params[2];
    
    // 计算网格和块大小
    int block_size = 256;
    int grid_size = (num_points + block_size - 1) / block_size;
    
    // 启动内核
    preprocessPointsKernel<<<grid_size, block_size, 0, stream>>>(
        points, num_points, voxel_features, voxel_num_points,
        min_x, min_y, min_z, max_x, max_y, max_z,
        voxel_x, voxel_y, voxel_z, grid_x, grid_y, grid_z,
        max_points_per_voxel, feature_num);
}

} // namespace tomato_slam
